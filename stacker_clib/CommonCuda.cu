#include <iostream>
#include "cuda_error.h"
#include "Chunk.h"
#include "CommonCuda.h"

void allocate_cuda_data(DataContainer& data, /*{{{*/
                        const size_t nchan, const size_t nstokes,
                        const size_t chunk_size)
{
    CudaSafeCall(hipMalloc( (void**)&data.u, sizeof(float)*chunk_size));
    CudaSafeCall(hipMalloc( (void**)&data.v, sizeof(float)*chunk_size));
    CudaSafeCall(hipMalloc( (void**)&data.w, sizeof(float)*chunk_size));
    CudaSafeCall(hipMalloc( (void**)&data.data_real, sizeof(float)*chunk_size*nchan*nstokes));
    CudaSafeCall(hipMalloc( (void**)&data.data_imag, sizeof(float)*chunk_size*nchan*nstokes));
    CudaSafeCall(hipMalloc( (void**)&data.data_weight, sizeof(float)*chunk_size*nstokes));
    CudaSafeCall(hipMalloc( (void**)&data.data_flag, sizeof(int)*chunk_size*nchan*nstokes));
    CudaSafeCall(hipMalloc( (void**)&data.spw, sizeof(int)*chunk_size));
    CudaSafeCall(hipMalloc( (void**)&data.field, sizeof(int)*chunk_size));
};/*}}}*/
void setup_freq(DataContainer& data, float* freq, const size_t nchan,/*{{{*/
                const size_t nspw)
{
    CudaSafeCall(hipMalloc( (void**)&data.freq, sizeof(float)*nchan*nspw));
    CudaSafeCall(hipMemcpy(data.freq, freq, sizeof(float)*nchan*nspw,
                hipMemcpyHostToDevice));
}/*}}}*/
void copy_data_to_cuda(DataContainer& data, Chunk& chunk)/*{{{*/
{
    size_t chunk_size = chunk.size();

    float* u = new float[chunk_size];
    float* v = new float[chunk_size];
    float* w = new float[chunk_size];
    int* spw = new int[chunk_size];
    int* field = new int[chunk_size];
    for(size_t uvrow = 0; uvrow < chunk_size; uvrow++)
    {
        u[uvrow] = chunk.inVis[uvrow].u;
        v[uvrow] = chunk.inVis[uvrow].v;
        w[uvrow] = chunk.inVis[uvrow].w;
        spw[uvrow] = chunk.inVis[uvrow].spw;
        field[uvrow] = chunk.inVis[uvrow].fieldID;
    }

    CudaSafeCall(hipMemcpy(data.u, u, sizeof(float)*chunk_size,
                hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(data.v, v, sizeof(float)*chunk_size,
                hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(data.w, w, sizeof(float)*chunk_size,
                hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(data.spw, spw, sizeof(float)*chunk_size,
                hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(data.field, field, sizeof(float)*chunk_size,
                hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(data.data_real,   chunk.data_real_in,
                sizeof(float)*chunk.size()*chunk.nChan()*chunk.nStokes(),
                hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(data.data_imag,   chunk.data_imag_in,
                sizeof(float)*chunk.size()*chunk.nChan()*chunk.nStokes(),
                hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(data.data_weight, chunk.weight_in,
                sizeof(float)*chunk.size()*chunk.nStokes(),
                hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(data.data_flag,   chunk.data_flag_in,
                sizeof(int)*chunk.size()*chunk.nChan()*chunk.nStokes(),
                hipMemcpyHostToDevice));

    delete[] u;
    delete[] v;
    delete[] w;
    delete[] spw;
    delete[] field;
}/*}}}*/
void copy_data_to_host(DataContainer& data, Chunk& chunk)/*{{{*/
{
    CudaSafeCall(hipMemcpy(chunk.data_real_out, data.data_real,
                sizeof(float)*chunk.size()*chunk.nChan()*chunk.nStokes(),
                hipMemcpyDeviceToHost));
    CudaSafeCall(hipMemcpy(chunk.data_imag_out, data.data_imag,
                sizeof(float)*chunk.size()*chunk.nChan()*chunk.nStokes(),
                hipMemcpyDeviceToHost));
    CudaSafeCall(hipMemcpy(chunk.weight_out, data.data_weight,
                sizeof(float)*chunk.size()*chunk.nStokes(),
                hipMemcpyDeviceToHost));
    CudaSafeCall(hipMemcpy(chunk.data_flag_out, data.data_flag,
                sizeof(int)*chunk.size()*chunk.nChan()*chunk.nStokes(),
                hipMemcpyDeviceToHost));
}/*}}}*/
